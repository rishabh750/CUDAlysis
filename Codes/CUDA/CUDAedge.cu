#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
#include <fstream>
#include <time.h>
using namespace std;
#define TILE 25
__global__ void
CUDAedge( int *a, const int b)
{
	int pos = threadIdx.x;
	if ((pos+1)%b!=0 && a[pos]!=a[pos+1])
		a[pos]=1;
	else
		a[pos]=0;
}
int main ()
{
	ifstream fin;
	fin.open("input.in");
	ofstream fout;
	fout.open("outputC.dat");
	int i,h,b,cs;
	double total=0,total1=0;
	fin>>cs;
	for (int k=1;k<=cs;k++)
	{
		clock_t st = clock();
		fin>>h>>b;
		int *ha,*da;
		int *r=new int[h*b];
		ha=new int[h*b];
		for ( i = 0 ; i<h*b ; i++ )
			fin>>ha[i];
		hipMalloc((void **) &da, h*b*sizeof (int));
		hipMemcpy(da,ha,h*b*sizeof(int),hipMemcpyHostToDevice);
		clock_t st1 = clock();
		CUDAedge <<<1,h*b>>> ( da , b) ;
		hipDeviceSynchronize();
		hipDeviceSynchronize();
		hipHostFree(ha);
		hipFree(da);
		clock_t et = clock();
		double t=double(et - st)/CLOCKS_PER_SEC;
		double t1=double(et - st1)/CLOCKS_PER_SEC;
		total+=t;
		total1+=t1;
		fout<<k<<'\t'<<h<<'\t'<<b<<'\t'<<h*b<<'\t'<<t<<'\t'<<t1<<endl;
		hipMemcpy(r,da,h*b*sizeof(int),hipMemcpyDeviceToHost);
	}
	//fout<<total<<'\t'<<total1;
	fin.close();
	fout.close();
}