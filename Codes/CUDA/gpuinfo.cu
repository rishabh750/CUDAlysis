#include "hip/hip_runtime.h"
#include ""
#include <iostream> 
#include <fstream> 
using namespace std;
int main()
{
	ofstream fout;
	fout.open("gpuinfo.dat");
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		fout<<"$Device Number:\t"<<i<<endl;
		fout<<"$Device name:\t"<<prop.name<<endl;
		fout<<"$Memory Clock Rate (KHz):\t"<<prop.memoryClockRate<<endl;
		fout<<"$Memory Bus Width (bits):\t"<<prop.memoryBusWidth<<endl;
		fout<<"$Peak Memory Bandwidth (GB/s):\t"<<2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6;
	}
	fout.close();
}